#include "hip/hip_runtime.h"
#include "utils.cuh"

__global__ void CudaRandomInit(uint64_t seed, hiprandState *state) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);
}