#include "hip/hip_runtime.h"
#include "triangle.cuh"

using namespace glm;

__host__ __device__ Triangle::Triangle(glm::vec3 p[], Material *material_ptr)
    : material_ptr_(material_ptr) {
  memcpy(p_, p, sizeof(glm::vec3) * 3);
}

__device__ bool Triangle::Hit(const Ray &ray, double t_from, double t_to,
                              HitRecord *out) {
  vec3 n = normalize(cross(p_[2] - p_[0], p_[1] - p_[2]));
  double den = dot(ray.direction(), n);
  double num = dot(p_[0] - ray.position(), n);
  double t = num / den;
  if (std::isnan(t) || std::isinf(t)) return false;
  if (t < t_from || t > t_to) return false;
  vec3 hit_point = ray.position() + (float)t * ray.direction();
  {
    double dot_values[3];
    for (int i = 0; i < 3; i++) {
      vec3 a = p_[i], b = p_[(i + 1) % 3];
      dot_values[i] = dot(cross(a - hit_point, b - hit_point), n);
    }
    for (int i = 0; i < 3; i++)
      if (dot_values[i] * dot_values[(i + 1) % 3] < 0) return false;
  }
  HitRecord record;
  record.t = t;
  record.normal = dot(n, ray.direction()) < 0 ? n : -n;
  record.material_ptr = material_ptr_;
  *out = record;
  return true;
}