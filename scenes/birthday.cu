#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION

#include <stb_image.h>

#include <cstdio>
#include <fstream>
#include <glm/glm.hpp>
#include <glm/gtc/constants.hpp>
#include <glm/gtx/rotate_vector.hpp>
#include <iostream>
#include <string>
#include <vector>

#include "camera.cuh"
#include "diffuse_light.cuh"
#include "hitable_list.cuh"
#include "lambertian.cuh"
#include "parallelepiped.cuh"
#include "parallelogram.cuh"
#include "ray_tracing.cuh"
#include "sky.cuh"
#include "sphere.cuh"
#include "textures/constant_texture.cuh"
#include "textures/image_texture.cuh"
#include "utils.cuh"

const int WIDTH = 1280, HEIGHT = 720;

hiprandState *d_states;

Camera *d_camera;
HitableList *d_world;
glm::vec3 *d_image;

struct DeviceImage {
  uint8_t *image;
  int height, width, components;
};

DeviceImage d_earthmap;

using glm::pi;
using glm::rotateX;
using glm::rotateY;
using glm::vec3;

__global__ void InitWorld(HitableList *world, Camera *camera,
                          DeviceImage earthmap) {
  new (world) HitableList();
  new (camera) Camera(vec3(278, 278, -800), vec3(278, 278, 0), vec3(0, 1, 0),
                      pi<double>() * 2 / 9, double(WIDTH) / HEIGHT);

  auto red_material_ptr = new Lambertian(vec3(0.65, 0.05, 0.05));
  auto white_material_ptr = new Lambertian(vec3(0.73, 0.73, 0.73));
  auto green_material_ptr = new Lambertian(vec3(0.12, 0.45, 0.15));

  auto light_material_ptr =
      new DiffuseLight(new ConstantTexture(vec3(1, 1, 1)));
  auto earthmap_texture_ptr = new ImageTexture(
      earthmap.height, earthmap.width, earthmap.components, earthmap.image);
  auto earthmap_material_ptr = new Lambertian(earthmap_texture_ptr);
  auto sky = new Sky();

  world->Append(sky);
  vec3 parallelograms[] = {
      vec3(0, 0, 0),       vec3(0, 555, 0),     vec3(0, 0, 555),
      vec3(555, 0, 0),     vec3(555, 555, 0),   vec3(555, 0, 555),
      vec3(213, 554, 332), vec3(213, 554, 227), vec3(343, 554, 332),
      vec3(0, 0, 0),       vec3(555, 0, 0),     vec3(0, 0, 555),
      vec3(555, 555, 555), vec3(0, 555, 555),   vec3(555, 0, 555),
      vec3(0, 555, 0),     vec3(555, 555, 0),   vec3(0, 555, 555)};
  world->Append(new Parallelogram(&parallelograms[0], red_material_ptr));
  world->Append(new Parallelogram(&parallelograms[3], green_material_ptr));
  world->Append(new Parallelogram(&parallelograms[6], light_material_ptr));
  world->Append(new Parallelogram(&parallelograms[9], white_material_ptr));
  world->Append(new Parallelogram(&parallelograms[12], white_material_ptr));
  world->Append(new Parallelogram(&parallelograms[15], white_material_ptr));

  world->Append(new Sphere(vec3(278, 278, 0), 100, earthmap_material_ptr));
}

void InitImageTextures(DeviceImage *earthmap) {
  auto data = stbi_load("resources/earthmap.jpg", &earthmap->width,
                        &earthmap->height, &earthmap->components, 0);
  int size = earthmap->height * earthmap->width * earthmap->components;
  hipMalloc(&earthmap->image, size);
  hipMemcpy(earthmap->image, data, size, hipMemcpyHostToDevice);
  auto err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);
}

int main() {
  Main(
      &d_states, &d_camera, &d_world, &d_image,
      [](HitableList *world, Camera *camera) {
        InitImageTextures(&d_earthmap);
        InitWorld<<<1, 1>>>(world, camera, d_earthmap);
      },
      HEIGHT, WIDTH, 200);
  return 0;
}