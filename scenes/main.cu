#include "hip/hip_runtime.h"
#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#include "camera.cuh"
#include "hitable_list.cuh"
#include "lambertian.cuh"
#include "ray_tracing.cuh"
#include "sky.cuh"
#include "sphere.cuh"
#include "textures/constant_texture.cuh"
#include "utils.cuh"

const int WIDTH = 1280, HEIGHT = 720;

hiprandState *d_states;

Camera *d_camera;
HitableList *d_world;
glm::vec3 *d_image;

void Output(std::vector<glm::vec3> &pixels, int height, int width) {
  std::string title = "image.ppm";
  LOG(INFO) << "Writing to " << title << "..." << std::endl;
  std::fstream fs(title, std::ios::out);
  fs << "P3\n" << width << " " << height << "\n255\n";
  for (int i = 0; i < width * height; i++) {
    auto color = pixels[i];
    for (int j = 0; j < 3; j++) fs << int(color[j] * 255) << " ";
  }
}

__global__ void InitWorld(HitableList *world, Camera *camera,
                          hiprandState *states) {
  auto const_tex = new ConstantTexture(glm::vec3(0, 1, 0));
  auto lambertian = new Lambertian(&states[0], const_tex);
  auto sphere_0 = new Sphere(glm::vec3(0, 0, -1), 0.5, lambertian);
  auto sphere_1 = new Sphere(glm::vec3(0, -100.5, -1), 100, lambertian);
  auto sky = new Sky();

  new (camera)
      Camera(glm::vec3(3, 3, 2), glm::vec3(0, 0, -1), glm::vec3(0, 1, 0),
             glm::radians(20), WIDTH * 1.0 / HEIGHT, &states[1]);

  new (world) HitableList();
  world->Append(sphere_0);
  world->Append(sphere_1);
  world->Append(sky);
}

int main() {
  hipError_t err;

  hipMalloc(&d_states, sizeof(hiprandState) * WIDTH * HEIGHT);
  hipMalloc(&d_image, sizeof(glm::vec3) * WIDTH * HEIGHT);
  hipMalloc(&d_world, sizeof(HitableList));
  hipMalloc(&d_camera, sizeof(Camera));
  err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  CudaRandomInit<<<WIDTH * HEIGHT / 64, 64>>>(10086, d_states);
  InitWorld<<<1, 1>>>(d_world, d_camera, d_states);
  hipDeviceSynchronize();
  err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  dim3 grid(HEIGHT / 8, WIDTH / 8);
  dim3 block(8, 8);
  RayTracing<<<grid, block>>>(d_world, d_camera, HEIGHT, WIDTH, 100, d_states,
                              d_image);
  hipDeviceSynchronize();
  err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  std::vector<glm::vec3> image(HEIGHT * WIDTH);
  err = hipMemcpy(image.data(), d_image, sizeof(glm::vec3) * HEIGHT * WIDTH,
                   hipMemcpyDeviceToHost);
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  Output(image, HEIGHT, WIDTH);
  return 0;
}