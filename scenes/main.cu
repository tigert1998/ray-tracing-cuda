#include "hip/hip_runtime.h"
#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#include "camera.cuh"
#include "hitable_list.cuh"
#include "lambertian.cuh"
#include "ray_tracing.cuh"
#include "sky.cuh"
#include "sphere.cuh"
#include "textures/constant_texture.cuh"
#include "utils.cuh"

const int WIDTH = 1280, HEIGHT = 720;

hiprandState *d_states;

Camera *d_camera;
HitableList *d_world;
glm::vec3 *d_image;

void Output(std::vector<glm::vec3> &pixels, int height, int width) {
  std::string title = "image.ppm";
  LOG(INFO) << "Writing to " << title << "..." << std::endl;
  std::fstream fs(title, std::ios::out);
  fs << "P3\n" << width << " " << height << "\n255\n";
  for (int i = 0; i < width * height; i++) {
    auto color = pixels[i];
    for (int j = 0; j < 3; j++) fs << int(color[j] * 255) << " ";
  }
}

__global__ void InitWorld(HitableList *world, Camera *camera) {
  auto green_tex = new ConstantTexture(glm::vec3(0, 1, 0));
  auto green_lambertian = new Lambertian(green_tex);
  auto red_tex = new ConstantTexture(glm::vec3(1, 0, 0));
  auto red_lambertian = new Lambertian(red_tex);

  auto sphere_0 = new Sphere(glm::vec3(0, 0, -1), 0.5, red_lambertian);
  auto sphere_1 = new Sphere(glm::vec3(0, -100.5, -1), 100, green_lambertian);
  auto sky = new Sky();

  new (camera)
      Camera(glm::vec3(0, 0, 0), glm::vec3(0, 0, -1), glm::vec3(0, 1, 0),
             glm::radians<float>(120), WIDTH * 1.0 / HEIGHT);

  new (world) HitableList();
  world->Append(sky);
  world->Append(sphere_0);
  world->Append(sphere_1);
}

int main() {
  hipError_t err;

  hipMalloc(&d_states, sizeof(hiprandState) * WIDTH * HEIGHT);
  hipMalloc(&d_image, sizeof(glm::vec3) * WIDTH * HEIGHT);
  hipMalloc(&d_world, sizeof(HitableList));
  hipMalloc(&d_camera, sizeof(Camera));
  err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  InitWorld<<<1, 1>>>(d_world, d_camera);
  CudaRandomInit<<<WIDTH * HEIGHT / 64, 64>>>(10086, d_states);
  hipDeviceSynchronize();
  err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    dim3 block(8, 8);
    dim3 grid((HEIGHT + block.x - 1) / block.x,
              (WIDTH + block.y - 1) / block.y);
    hipEventRecord(start);
    RayTracing<<<grid, block>>>(d_world, d_camera, HEIGHT, WIDTH, 100, d_states,
                                d_image);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    err = hipGetLastError();
    CHECK(err == hipSuccess) << hipGetErrorString(err);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    LOG(INFO) << "Ray tracing finished in " << ms << "ms.";
  }

  std::vector<glm::vec3> image(HEIGHT * WIDTH);
  err = hipMemcpy(image.data(), d_image, sizeof(glm::vec3) * HEIGHT * WIDTH,
                   hipMemcpyDeviceToHost);
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  Output(image, HEIGHT, WIDTH);
  return 0;
}