#include "hip/hip_runtime.h"
#include <assimp/cimport.h>
#include <assimp/postprocess.h>
#include <assimp/scene.h>

#include <cstdio>
#include <fstream>
#include <glm/glm.hpp>
#include <glm/gtc/constants.hpp>
#include <glm/gtx/rotate_vector.hpp>
#include <iostream>
#include <string>
#include <vector>

#include "bvh.cuh"
#include "camera.cuh"
#include "dielectric.cuh"
#include "diffuse_light.cuh"
#include "hitable_list.cuh"
#include "lambertian.cuh"
#include "metal.cuh"
#include "parallelepiped.cuh"
#include "parallelogram.cuh"
#include "ray_tracing.cuh"
#include "sky.cuh"
#include "sphere.cuh"
#include "textures/constant_texture.cuh"
#include "utils.cuh"

const int WIDTH = 1280, HEIGHT = 720;

hiprandState *d_states;

Camera *d_camera;
HitableList *d_world;
glm::vec3 *d_image;
Face *d_faces;

using glm::pi;
using glm::vec3;

__global__ void InitWorld(HitableList *world, Camera *camera) {
  new (world) HitableList();
  new (camera)
      Camera(vec3(-0.025, 0.1, -0.5), vec3(-0.025, 0.1, 0), vec3(0, 1, 0),
             pi<double>() * 2 / 9, double(WIDTH) / HEIGHT);
  auto sky = new Sky();
  world->Append(sky);
}

__global__ void InitModel(HitableList *world, Face *faces, int n) {
  auto white_material_ptr = new Metal(vec3(0.72, 0.72, 0.72));
  auto green_material_ptr = new Lambertian(vec3(0.12, 0.45, 0.15));
  vec3 parallelograms[] = {vec3(-0.025 - 0.5, 0.1 - 0.5, 1.2),
                           vec3(-0.025 + 0.5, 0.1 - 0.5, 1.2),
                           vec3(-0.025 - 0.5, 0.1 + 0.5, 1.2)};
  world->Append(new Parallelogram(&parallelograms[0], green_material_ptr));
  auto bvh = new BVH(faces, n, white_material_ptr);
  world->Append(bvh);
}

void ImportModel(const std::string &path) {
  const aiScene *scene = aiImportFile(
      path.c_str(), aiProcess_GlobalScale | aiProcess_CalcTangentSpace |
                        aiProcess_Triangulate);
  std::vector<Face> faces;
  for (int i = 0; i < scene->mNumMeshes; i++) {
    auto mesh = scene->mMeshes[i];
    faces.reserve(faces.capacity() + mesh->mNumFaces);
    for (int j = 0; j < mesh->mNumFaces; j++) {
      Face face;
      for (int k = 0; k < 3; k++) {
        int idx = mesh->mFaces[j].mIndices[k];
        auto vertex = mesh->mVertices[idx];
        face.points[k] = vec3(vertex.x, vertex.y, vertex.z);
      }
      faces.emplace_back(face);
    }
  }
  aiReleaseImport(scene);
  hipMalloc(&d_faces, sizeof(Face) * faces.size());
  hipMemcpy(d_faces, faces.data(), sizeof(Face) * faces.size(),
             hipMemcpyHostToDevice);
  InitModel<<<1, 1>>>(d_world, d_faces, faces.size());
  hipDeviceSynchronize();
  auto err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);
}

int main() {
  hipError_t err;

  hipMalloc(&d_states, sizeof(hiprandState) * WIDTH * HEIGHT);
  hipMalloc(&d_image, sizeof(glm::vec3) * WIDTH * HEIGHT);
  hipMalloc(&d_world, sizeof(HitableList));
  hipMalloc(&d_camera, sizeof(Camera));
  err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  InitWorld<<<1, 1>>>(d_world, d_camera);
  CudaRandomInit<<<WIDTH * HEIGHT / 64, 64>>>(10086, d_states);
  hipDeviceSynchronize();
  err = hipGetLastError();
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  ImportModel("bunny.obj");

  {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    dim3 block(8, 8);
    dim3 grid((HEIGHT + block.x - 1) / block.x,
              (WIDTH + block.y - 1) / block.y);
    hipEventRecord(start);
    RayTracing<<<grid, block>>>(d_world, d_camera, HEIGHT, WIDTH, 20, d_states,
                                d_image);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    err = hipGetLastError();
    CHECK(err == hipSuccess) << hipGetErrorString(err);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    LOG(INFO) << "Ray tracing finished in " << ms << "ms.";
  }

  std::vector<glm::vec3> image(HEIGHT * WIDTH);
  err = hipMemcpy(image.data(), d_image, sizeof(glm::vec3) * HEIGHT * WIDTH,
                   hipMemcpyDeviceToHost);
  CHECK(err == hipSuccess) << hipGetErrorString(err);

  WriteImage(image, HEIGHT, WIDTH, "image.jpeg");
  return 0;
}